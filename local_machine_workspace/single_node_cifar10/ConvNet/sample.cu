#include "hip/hip_runtime.h"
//#include <hip/hip_runtime.h>
//#include "hip/hip_runtime.h"
//#include ""

__global__ void func(float *d_pnt) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  //d_pnt[idx % 10]++;
  //atomicAdd(&d_pnt[idx % 10], 1);
}

void foo(float *d_pnt)
{
  dim3 block_dim(1, 1, 1);
  dim3 grid_dim(10000, 1, 1);
  
  func <<< 1, 1 >>> (d_pnt);
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
} 

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
